#include <mpi.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include "commdom.hpp"

//#include "shrUtils.h"
// utilities and system includes
// // CUDA-C includes
#include "hip/hip_runtime.h"
#define BUFSIZE 256
#define TAG 0
//  
int devCount;
int myid;
int ihavecuda;
int nodes[256];
int nocuda[256];
int deviceselector=0;
//     
string IAM = "MMMESH";

int main(int argc, char *argv[])
{
  char idstr[256];
  char idstr2[256];
  char buff[BUFSIZE];
  int i;
  int numprocs, rank, namelen;
  char processor_name[MPI_MAX_PROCESSOR_NAME];
  freopen("/dev/null", "w", stderr); /* Hide errors from nodes with no CUDA cards */

  MPI_Status stat;
  MPI_Init(NULL, NULL);
  MPI_Comm  world_comm = MPI_COMM_WORLD;

  string  namei = "";
  if(argc==2) namei = argv[1];

  // CommDom
  CommDom  CD = CommDom();
  CD.init();
  CD.set_app_type(IAM);
  CD.set_world_comm(world_comm);
  CD.set_app_name(namei);

  int  app_id = -1;
  int  n_apps = -1;
  MPI_Comm  local_comm;
  local_comm = CD.set_mpi_comms(); 
  MPI_Barrier(local_comm);

  int local_rank = -1;
  MPI_Comm_rank(local_comm, &local_rank);

/*
  MPI_Comm_size(local_comm, &numprocs);
  MPI_Comm_rank(local_comm, &rank);
  MPI_Get_processor_name(processor_name, &namelen);
  MPI_Comm_rank(local_comm, &myid);
  if (myid == 0)
  {
    printf("  We have %d processors\n", numprocs);
    printf("  Spawning from \'%s.%s\' \n", processor_name, namei.c_str());
    printf("  CUDA MPI\n");
    printf("\n");
    for(i=1; i<numprocs;i++)
    {
      buff[0]='\0';
      MPI_Send(buff, BUFSIZE, MPI_CHAR, i, TAG, local_comm);
    }
    printf("\n\n\n");
    printf("  Probing nodes...\n");
    printf("     Node        Psid  CUDA Cards (devID)\n");
    printf("     ----------- ----- ---- ----------\n");
    for(i=1; i<numprocs;i++)
    {
      MPI_Recv(buff, BUFSIZE, MPI_CHAR, i, TAG, local_comm, &stat);
      printf("%s\n", buff);
    }
    printf("\n");
    MPI_Finalize(); 
  }
  else
  {
   MPI_Recv(buff, BUFSIZE, MPI_CHAR, 0, TAG, local_comm, &stat);
   MPI_Get_processor_name(processor_name, &namelen);
   hipGetDeviceCount(&devCount);
   buff[0]='\0';
   idstr[0]='\0';
   if (devCount == 0) 
   {
     sprintf(idstr,"- %-11s %5d %4d NONE", processor_name, rank, devCount);
     ihavecuda=0;
   }
   else
   {
     ihavecuda=1;
     if (devCount >= 1)
     {
       sprintf(idstr, "+ '%s'.%-11s %5d %4d", namei.c_str(), processor_name, rank, devCount);
       idstr2[0]='\0';
       for (int i = 0; i < devCount; ++i)
       {
         hipDeviceProp_t devProp;
         hipGetDeviceProperties(&devProp, i);
         sprintf(idstr2, " %s (%d) ", devProp.name, i);
         strncat(idstr,idstr2,BUFSIZE);
       }
     }
     else
     {
       hipDeviceProp_t devProp;
       hipGetDeviceProperties(&devProp, i);
       sprintf(idstr, "%-11s %5d %4d %s", processor_name, rank, devCount, devProp.name);
     } 
   } 
   strncat(buff, idstr, BUFSIZE);
   MPI_Send(buff, BUFSIZE, MPI_CHAR, 0, TAG, local_comm);
  }
*/

  int n_send = 1;
  int n_recv = 1;
  double send[1];
  double recv[1];

  send[0] =  1e20;
  recv[0] = -1e20;

  string  namej = "";
  if(namei=="SOLID")
  {
    namej   = "FLUID";
    send[0] = 1.0; 
  }
  if(namei=="FLUID")
  {
    namej   = "SOLID";
    send[0] = -1.0;  
  }

  MPI_Comm  commij;
  CD.get_commij(namej, &commij);
  CD.__mpi_sendrecv_real__(send, n_send, recv, n_recv, local_comm, commij); 

  MPI_Barrier(commij);
  if(local_rank==0) cout<<" ===>'"<< namei <<"': "<< recv[0] <<"<=== \n"; 

  CD.locator_create2(local_comm, commij, 1e-3); 

/*
  int        n_vertices_j = 0 
  double *vertex_coords_j = NULL; 

  int     n_vertices_i    = 0 
  int     n_elements_i    = 0
  double *vertex_coords_i = NULL; 
  int    *vertex_num_i    = NULL;  

  CD.locator_set_mesh(n_vertices_i, 
                      n_elements_i, 
                      vertex_coords_i, 
                      vertex_num_i,
                      n_vertices_j, 
                      vertex_coords_j)

  CD.save_dist_coords(local_rank); 
*/  

  MPI_Finalize();
  return 0;
} 

